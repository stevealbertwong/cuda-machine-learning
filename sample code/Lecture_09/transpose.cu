#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <assert.h>
#include <vector>
#include "utils.h"

const unsigned warp_size = 32;

__global__
void simpleTranspose(int* array_in, int* array_out, size_t n_rows, size_t n_cols) {
  const size_t tid = threadIdx.x + blockDim.x * blockIdx.x;

  size_t col = tid % n_cols;
  size_t row = tid / n_cols;

  if(col < n_cols && row < n_rows) {
    array_out[col * n_rows + row] = array_in[row * n_cols + col];
  }
}

__global__
void simpleTranspose2D(int* array_in, int* array_out, size_t n_rows, size_t n_cols) {
  const size_t col = threadIdx.x + blockDim.x * blockIdx.x;
  const size_t row = threadIdx.y + blockDim.y * blockIdx.y;

  if(col < n_cols && row < n_rows) {
    array_out[col * n_rows + row] = array_in[row * n_cols + col];
  }
}

template<int num_warps>
__global__
void fastTranspose(int* array_in, int* array_out, size_t n_rows, size_t n_cols) {
  const int warp_id  = threadIdx.y;
  const int lane     = threadIdx.x;

  __shared__ int block[warp_size][warp_size+1];

  const int bc = blockIdx.x;
  const int br = blockIdx.y;

  // Load 32x32 block into shared memory
  size_t gc = bc * warp_size + lane; // Global column index

  size_t gr;

  for(int i = 0; i < warp_size / num_warps; ++i) {
    gr = br * warp_size + i * num_warps + warp_id; // Global row index
    block[i * num_warps + warp_id][lane] = array_in[gr * n_cols + gc];
  }

  __syncthreads();

  // Now we switch to each warp outputting a row, which will read
  // from a column in the shared memory. This way everything remains
  // coalesced.
  gr = br * warp_size + lane;

  for(int i = 0; i < warp_size / num_warps; ++i) {
    gc = bc * warp_size + i * num_warps + warp_id;
    array_out[gc * n_rows + gr] = block[lane][i * num_warps + warp_id];
  }
}

void isTranspose(const std::vector<int>& A,
                 const std::vector<int>& B,
                 size_t n) {
  for(size_t i = 0; i < n; ++i) {
    for(size_t j = 0; j < n; ++j) {
      assert(A[n * i + j] == B[n * j + i]);
    }
  }
}

void print_out(int n_iter, size_t n, float elapsed) {
  printf("GPU took %g ms\n",elapsed / n_iter);
  printf("Effective bandwidth is %g GB/s\n",
  (2*sizeof(int)*n*n*n_iter)/(1e9*1e-3*elapsed));
}

#define MEMCOPY_ITERATIONS 10

int main(void) {
  const size_t n = (1<<15);

  printf("Number of MB to transpose: %ld\n\n",sizeof(int) * n * n / 1024 / 1024);

  int num_threads, num_blocks;

  std::vector<int> h_in(n * n);
  std::vector<int> h_out(n * n);

  for(size_t i = 0; i < n * n; ++i) {
    h_in[i] = random() % 100;
  }

  int* d_in, *d_out;
  checkCudaErrors(hipMalloc(&d_in,  sizeof(int) * n * n));
  checkCudaErrors(hipMalloc(&d_out, sizeof(int) * n * n));

  GpuTimer timer;
  timer.start();
  for(unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
    checkCudaErrors(hipMemcpy(d_out, d_in, sizeof(int) * n * n,
			       hipMemcpyDeviceToDevice));
  }
  timer.stop();
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  printf("Bandwidth bench\n");
  print_out(MEMCOPY_ITERATIONS,n,timer.elapsed());

  checkCudaErrors(hipMemcpy(d_in, &h_in[0], sizeof(int) * n * n,
			     hipMemcpyHostToDevice));

  num_threads = 256;
  num_blocks = (n * n + num_threads - 1) / num_threads;

  timer.start();
  for(unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
    simpleTranspose<<<num_blocks, num_threads>>>(d_in, d_out, n, n);
  }
  timer.stop();

  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  for(size_t i = 0; i < n * n; ++i) {
    h_out[i] = -1;
  }
  checkCudaErrors(hipMemcpy(&h_out[0], d_out, sizeof(int) * n * n,
			     hipMemcpyDeviceToHost));

  isTranspose(h_in, h_out, n);

  printf("\nsimpleTranspose\n");
  print_out(MEMCOPY_ITERATIONS,n,timer.elapsed());

  dim3 block_dim(8, 32);
  dim3 grid_dim(n / 8, n / 32);

  timer.start();
  for(unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
    simpleTranspose2D<<<grid_dim, block_dim>>>(d_in, d_out, n, n);
  }
  timer.stop();
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  for(size_t i = 0; i < n * n; ++i) {
    h_out[i] = -1;
  }    
  checkCudaErrors(hipMemcpy(&h_out[0], d_out, sizeof(int) * n * n,
			     hipMemcpyDeviceToHost));
         
  isTranspose(h_in, h_out, n);

  printf("\nsimpleTranspose2D\n");
  print_out(MEMCOPY_ITERATIONS,n,timer.elapsed());

  const int num_warps_per_block = 256/32;
  assert(warp_size % num_warps_per_block == 0);
  block_dim.x = warp_size;
  block_dim.y = num_warps_per_block;
  grid_dim.x = n / warp_size;
  grid_dim.y = n / warp_size;

  timer.start();
  for(unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
    fastTranspose<num_warps_per_block><<<grid_dim, block_dim>>>(d_in, d_out, n,
								n);
  }
  timer.stop();
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  for(size_t i = 0; i < n * n; ++i) {
    h_out[i] = -1;
  }     
  checkCudaErrors(hipMemcpy(&h_out[0], d_out, sizeof(int) * n * n,
			     hipMemcpyDeviceToHost));
                 
  isTranspose(h_in, h_out, n);

  printf("\nfastTranspose\n");
  print_out(MEMCOPY_ITERATIONS,n,timer.elapsed());

  checkCudaErrors(hipFree(d_in));
  checkCudaErrors(hipFree(d_out));

  return 0;
}
