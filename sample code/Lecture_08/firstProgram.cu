#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <utils.h>

using std::vector;

__device__ __host__
int f(int i) {
    return i*i;
}

__global__
void kernel(int* out) {
    out[threadIdx.x] = f(threadIdx.x);
}

int main(int argc, const char** argv) {
    int N = 32;

    if (checkCmdLineFlag(argc, argv, "N")) {
        N = getCmdLineArgumentInt(argc, argv, "N");
        printf("Using %d threads = %d warps\n",N, (N+31)/32);   
    }     

    int* d_output;

    /* checkCudaErrors:
       A wrapper function we wrote to test whether an error occurred
       when launching a kernel.
       hipMalloc:
       Allocated memory on device
       */
    checkCudaErrors(hipMalloc(&d_output, sizeof(int) * N));

    /* This is like a parallel for loop.
       kernel is the function above.
       d_output is the input variable.
       This call will execute the function kernel using N threads.
       Each thread gets a different threadIdx.x value.
       */
    kernel<<<1, N>>>(d_output);

    /* This is just to check that the kernel executed as expected. */
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    vector<int> h_output(N);
    /* This function copies the data back from GPU to CPU.
       See hipMemcpyDeviceToHost
       You also have
       hipMemcpyHostToDevice
       */
    checkCudaErrors(hipMemcpy(&h_output[0], d_output, sizeof(int) * N,
                               hipMemcpyDeviceToHost));

    for(int i = 0; i < N; ++i) {
        if (i==0 || i==N-1 || i%(N/10) == 0)
		printf("Entry %10d, written by thread %5d\n", h_output[i], i);
        assert(h_output[i] == f(i));
    }

    /* Free memory on the device. */
    checkCudaErrors(hipFree(d_output));

    return 0;
}